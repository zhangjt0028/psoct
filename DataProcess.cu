#include "hip/hip_runtime.h"
#include<DataProcess.h>
#include<hipfft/hipfft.h>

DataProcess::DataProcess(){
    MAX_NUM_SAVED_VOLUMEM_IN_MEMORY  = 800;
    MAX_NUM_SAVED_VOLUMEM_IN_MEMORY3D = 3200;
    m_sampleLength = 1152;
    captureflag = 1;//0表示该状态不进行采集
    scanMode = 2; //2 or 3

    //hammingwindow[i] = 0.54-0.46*cos(2 * 3.1415926 * i / mainWidget::m_sampleLength);

    double turki[1152] = {0,0.00011919,0.00047672,0.0010724,0.001906,0.002977,0.004285,0.0058293,0.0076093,0.009624,0.011873,0.014354,0.017067,0.02001,0.023181,0.026581,0.030206,0.034054,0.038126,0.042417,0.046926,0.051652,0.056591,0.061742,0.067101,0.072667,0.078437,0.084407,0.090576,0.09694,0.1035,0.11024,0.11717,0.12429,0.13158,0.13905,0.14669,0.1545,0.16247,0.17061,0.1789,0.18734,0.19594,0.20467,0.21355,0.22257,0.23172,0.24099,0.25039,0.25991,0.26955,0.27929,0.28914,0.29909,0.30913,0.31927,0.32949,0.33979,0.35017,0.36062,0.37114,0.38172,0.39235,0.40304,0.41377,0.42455,0.43536,0.4462,0.45706,0.46795,0.47885,0.48977,0.50068,0.5116,0.52251,0.53341,0.5443,0.55516,0.566,0.5768,0.58757,0.5983,0.60898,0.61961,0.63018,0.64069,0.65113,0.6615,0.67179,0.682,0.69213,0.70216,0.7121,0.72193,0.73166,0.74128,0.75079,0.76017,0.76943,0.77856,0.78756,0.79643,0.80515,0.81372,0.82215,0.83042,0.83854,0.84649,0.85428,0.8619,0.86934,0.87661,0.8837,0.89061,0.89733,0.90387,0.91021,0.91635,0.9223,0.92804,0.93358,0.93891,0.94404,0.94895,0.95365,0.95813,0.9624,0.96644,0.97026,0.97386,0.97723,0.98037,0.98329,0.98597,0.98842,0.99064,0.99263,0.99438,0.99589,0.99717,0.99821,0.99902,0.99958,0.99991,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0.99991,0.99958,0.99902,0.99821,0.99717,0.99589,0.99438,0.99263,0.99064,0.98842,0.98597,0.98329,0.98037,0.97723,0.97386,0.97026,0.96644,0.9624,0.95813,0.95365,0.94895,0.94404,0.93891,0.93358,0.92804,0.9223,0.91635,0.91021,0.90387,0.89733,0.89061,0.8837,0.87661,0.86934,0.8619,0.85428,0.84649,0.83854,0.83042,0.82215,0.81372,0.80515,0.79643,0.78756,0.77856,0.76943,0.76017,0.75079,0.74128,0.73166,0.72193,0.7121,0.70216,0.69213,0.682,0.67179,0.6615,0.65113,0.64069,0.63018,0.61961,0.60898,0.5983,0.58757,0.5768,0.566,0.55516,0.5443,0.53341,0.52251,0.5116,0.50068,0.48977,0.47885,0.46795,0.45706,0.4462,0.43536,0.42455,0.41377,0.40304,0.39235,0.38172,0.37114,0.36062,0.35017,0.33979,0.32949,0.31927,0.30913,0.29909,0.28914,0.27929,0.26955,0.25991,0.25039,0.24099,0.23172,0.22257,0.21355,0.20467,0.19594,0.18734,0.1789,0.17061,0.16247,0.1545,0.14669,0.13905,0.13158,0.12429,0.11717,0.11024,0.1035,0.09694,0.090576,0.084407,0.078437,0.072667,0.067101,0.061742,0.056591,0.051652,0.046926,0.042417,0.038126,0.034054,0.030206,0.026581,0.023181,0.02001,0.017067,0.014354,0.011873,0.009624,0.0076093,0.0058293,0.004285,0.002977,0.001906,0.0010724,0.00047672,0.00011919,0};
    for (int i = 0; i < m_sampleLength; ++i)
    {
        hammingwindow[i] = turki[i];
    }

    //初始化本底///////////////
    U16 *curData = ssoctThread->m_volumeMemBuffer[availbleIndex];

    for (int j = 0; j < MAX_NUM_SAVED_VOLUMEM_IN_MEMORY; j++)
    {
        for (int i = 0; i < m_sampleLength; i++)
        {
            m_BG[i] += ((curData[j * m_sampleLength + i ] >> 4) - 2048);
        }
    }
    for (int i = 0; i < m_sampleLength; ++i)
    {
        m_BG[i] /= MAX_NUM_SAVED_VOLUMEM_IN_MEMORY;
    }
    /////////////////////////
}



void DataProcess::fftBscan(float* numbers)
{
//    if (m_cosphy)
//            mkl_free(m_cosphy);
    m_cosphy = (float *)mkl_malloc(m_sampleLength * sizeof(float), 64);
//    if (m_sinphy)
//            mkl_free(m_sinphy);
    m_sinphy = (float *)mkl_malloc(m_sampleLength * sizeof(float), 64);


   
    
    /////////////////////////////////////
    //qDebug()<<"time1 "<<QTime::currentTime();

    int TEMP_BSCANS_LINES = 800;
    int TEMP_CSCANS_LINES = 1;

    const int numel = m_sampleLength * TEMP_BSCANS_LINES * TEMP_CSCANS_LINES;



    MKL_Complex8* m_Cdata; // fft中的复数data
    m_Cdata = (MKL_Complex8*)mkl_malloc(m_sampleLength * TEMP_BSCANS_LINES * TEMP_CSCANS_LINES
        * sizeof(MKL_Complex8), 32);

    MKL_Complex8* m_Cdata1; // fft中的复数data
    m_Cdata1 = (MKL_Complex8*)mkl_malloc(m_sampleLength * TEMP_BSCANS_LINES * TEMP_CSCANS_LINES
        * sizeof(MKL_Complex8), 32);
    float tmp, phy;
    double m_dispersionW0 = ui->doubleSpinBoxw0->text().toDouble();//添加数据
    double m_dispersionA1 = ui->doubleSpinBoxa1->text().toDouble();//添加数据
    double m_dispersionA2 = ui->doubleSpinBoxa2->text().toDouble();//添加数据
    for (unsigned int i = 0; i < mainWidget::m_sampleLength; ++i)
    {
        tmp = (i - m_dispersionW0) * (i - m_dispersionW0);
        phy = m_dispersionA1 * tmp / 10000.0 + m_dispersionA2 * tmp * (i - m_dispersionW0) / 100000000.0;
        m_cosphy[i] = cos(phy);
        m_sinphy[i] = sin(phy);
    }
    
//    // 整块运算，对插值后的数据加入色散补偿，存入m_Cdata中。
//    for (unsigned int j = 0; j < 1; ++j)
//    {
//        for (unsigned int i = 0; i < mainWidget::m_sampleLength; ++i)
//        {
//            m_Cdata[j * mainWidget::m_sampleLength + i].real =
//                numbers[j * mainWidget::m_sampleLength + i] * m_cosphy[i];
//            m_Cdata[j * mainWidget::m_sampleLength + i].imag =
//                numbers[j * mainWidget::m_sampleLength + i] * m_sinphy[i];
//        }
//    }


    //qDebug()<<"m_cosphy"<<m_cosphy[50];
    //CalcDispersionParameter();
    for (unsigned int j = 0; j < TEMP_BSCANS_LINES; ++j)
    {
        for (unsigned int i = 0; i < m_sampleLength; ++i)
        {
            m_Cdata[j * m_sampleLength + i].real =
                numbers[j * m_sampleLength + i] * m_cosphy[i];
            m_Cdata[j * m_sampleLength + i].imag =
                numbers[j * m_sampleLength + i] * m_sinphy[i];
        }
    }

    
 
    
//    // 初始化mkl的fft命令;
    MKL_LONG status;
    DFTI_DESCRIPTOR_HANDLE m_FFThandle; // FFT handle

    status = DftiCreateDescriptor(&m_FFThandle, DFTI_SINGLE,
        DFTI_COMPLEX, 1, mainWidget::m_sampleLength);
    status = DftiSetValue(m_FFThandle, DFTI_PLACEMENT, DFTI_NOT_INPLACE);
    status = DftiSetValue(m_FFThandle, DFTI_NUMBER_OF_TRANSFORMS, TEMP_BSCANS_LINES * TEMP_CSCANS_LINES);
    status = DftiSetValue(m_FFThandle, DFTI_INPUT_DISTANCE, mainWidget::m_sampleLength);
    status = DftiSetValue(m_FFThandle, DFTI_OUTPUT_DISTANCE, mainWidget::m_sampleLength);
    status = DftiCommitDescriptor(m_FFThandle);
//    status = DftiCreateDescriptor(&m_FFThandle, DFTI_SINGLE,
//        DFTI_COMPLEX, 1, mainWidget::m_sampleLength);
//    status = DftiSetValue(m_FFThandle, DFTI_NUMBER_OF_TRANSFORMS, 1);
//    status = DftiSetValue(m_FFThandle, DFTI_INPUT_DISTANCE, mainWidget::m_sampleLength);
//    status = DftiCommitDescriptor(m_FFThandle);

    // 整块FFT
    DftiComputeForward(m_FFThandle, m_Cdata, m_Cdata1);

    status = DftiFreeDescriptor(&m_FFThandle);

    // 整块取模，结果保存在numbers中
    vcAbs(mainWidget::m_sampleLength * TEMP_BSCANS_LINES * TEMP_CSCANS_LINES, m_Cdata1, numbers);


    float* numberslog;
    numberslog = (float*)mkl_malloc(mainWidget::m_sampleLength * TEMP_BSCANS_LINES * TEMP_CSCANS_LINES
        * sizeof(float), 32);


    vslog10(&numel, numbers, numberslog);

    float alpha = 20;
    const int incx = 1;
    cblas_sscal(numel, alpha, numberslog, incx);

    mkl_free(m_Cdata);
    mkl_free(m_Cdata1);
    //mkl_free(numbers);

    // 开始考虑绘图
    plotBscan(numberslog);

    mkl_free(numberslog);
    mkl_free(m_cosphy);
    mkl_free(m_sinphy);
////    //qDebug()<<"time2 "<<QTime::currentTime();
}